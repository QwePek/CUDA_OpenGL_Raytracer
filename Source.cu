#include "hip/hip_runtime.h"
#include "pch.h"
#include "GL/glew.h"
#include <GLFW/glfw3.h>

#include <imgui/imgui_impl_glfw.h>
#include <imgui/imgui_impl_opengl3.h>

#include "src/Rendering/Renderer.h"
#include "src/Rendering/VertexBuffer.h"
#include "src/Rendering/VertexBufferLayout.h"
#include "src/Rendering/IndexBuffer.h"
#include "src/Rendering/Texture.h"
#include "src/Utils/Utils.h"
#include "src/Raytracing/HittableList.h"
#include "src/Raytracing/Objects/Sphere.h"
#include "src/Camera.h"
#include "src/Raytracing/Materials/Lambertian.h"
#include "src/Raytracing/Materials/Metal.h"
#include "src/Raytracing/Materials/Dielectric.h"

//CUDA
#include <hiprand/hiprand_kernel.h>
#include <>

float deltaTime = 0.0f;
float lastFrame = 0.0f;

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA Error: %s\n", hipGetErrorString(err));
        }
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__global__ void render_init(glm::u32vec2 imgSize, hiprandState* rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if ((i >= imgSize.x) || (j >= imgSize.y))
        return;
    int pixel_index = j * imgSize.x + i;
    hiprand_init(1984 + pixel_index, 0, 0, &rand_state[pixel_index]);
}

__global__ void render(dataPixels* data, glm::u32vec2 imgSize, Camera** cam, Hittable** world, hiprandState* rand_state)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if ((i >= imgSize.x) || (j >= imgSize.y))
        return;
    
    int pixelIndex = i + j * imgSize.x;
    hiprandState localRandState = rand_state[pixelIndex];
    glm::vec3 pixelColor(0.0f, 0.0f, 0.0f);
    for (int sampleIdx = 0; sampleIdx < (*cam)->getPerPixelSamples(); sampleIdx++){
        Ray r = (*cam)->getRay(i, j, &localRandState);
        pixelColor += (*cam)->rayColor(r, (*cam)->getMaxRecursionDepth(), world, &localRandState);
    }
    rand_state[pixelIndex] = localRandState;
    data[pixelIndex] = (*cam)->convertColor((*cam)->getPixelSampleScale() * pixelColor);
}

__global__ void rand_init(hiprandState* rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(1984, 0, 0, rand_state);
    }
}

#define RND (hiprand_uniform(&local_rand_state))

__global__ void initCamera(Camera** camera, glm::u32vec2 imgSize)
{
    *camera = new Camera(glm::vec3(13, 2, 3), glm::vec3(0, 0, 0), glm::vec3(0, 1, 0), 20, 10.0, 0.6, 16.0f / 9.0f, imgSize.x, 100, 50);
}

__global__ void initWorld(Hittable** worldObjects, Hittable** listObjects, hiprandState* rand_state)
{
    if (threadIdx.x != 0 || blockIdx.x != 0)
        return;

    int id = 0;
    hiprandState local_rand_state = *rand_state;
    listObjects[id++] = new Sphere(glm::vec3(0, -1000.0f, -1), 1000.0f, new Materials::Lambertian(glm::vec3(0.5f, 0.5f, 0.5f)));

    for (int a = -11; a < 11; a++) {
        for (int b = -11; b < 11; b++) {
            double choose_mat = Utils::generateRandomNumber(&local_rand_state);
            glm::vec3 center(a + 0.9 * Utils::generateRandomNumber(&local_rand_state), 0.2, b + 0.9 * Utils::generateRandomNumber(&local_rand_state));

            if ((center - glm::vec3(4, 0.2, 0)).length() > 0.9) {
                Material* sphere_material;

                if (choose_mat < 0.8) {
                    glm::vec3 albedo = Utils::Vector::randomVector(0.0, 1.0, &local_rand_state) * Utils::Vector::randomVector(0.0, 1.0, &local_rand_state);
                    sphere_material = new Materials::Lambertian(albedo);
                }
                else if (choose_mat < 0.95) {
                    glm::vec3 albedo = Utils::Vector::randomVector(0.5, 1.0, &local_rand_state);
                    double fuzz = Utils::generateRandomNumber(0, 0.5, &local_rand_state);
                    sphere_material = new Materials::Metal(albedo, fuzz);
                }
                else
                    sphere_material = new Materials::Dielectric(1.5);

                listObjects[id++] = new Sphere(center, 0.2, sphere_material);
            }
        }
    }
    listObjects[id++] = new Sphere(glm::vec3(0, 1, 0), 1.0, new Materials::Dielectric(1.5f));
    listObjects[id++] = new Sphere(glm::vec3(-4, 1, 0), 1.0, new Materials::Lambertian(glm::vec3(0.4f, 0.2f, 0.1f)));
    listObjects[id++] = new Sphere(glm::vec3(4, 1, 0), 1.0, new Materials::Metal(glm::vec3(0.7f, 0.6f, 0.5f), 0.0f));
    *rand_state = local_rand_state;
    *worldObjects = new HittableList(listObjects, 22 * 22 + 1 + 3);
}

__global__ void freeWorld(Hittable** worldObjects, Camera** camera) {
    delete *worldObjects;
    delete *camera;
}

void processInput(GLFWwindow* window)
{
    if (glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS)
        glfwSetWindowShouldClose(window, true);

    if (glfwGetKey(window, GLFW_KEY_X) == GLFW_PRESS)
        glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);
    else
        glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
}

void framebuffer_size_callback(GLFWwindow* window, int width, int height)
{
    glViewport(0, 0, width, height);
}

int main()
{
    if (!glfwInit())
    {
        std::cout << "Failed to initialize glfwInit()" << std::endl;
        return -1;
    }

    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

    //Essential window calculations
    hiprandState* curRandState1; //For world creation
    checkCudaErrors(hipMalloc((void**)&curRandState1, 1 * sizeof(hiprandState)));
    rand_init<<<1, 1>>>(curRandState1);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    glm::u32vec2 imgTmp(1920, 1920);
    Camera** cam;
    checkCudaErrors(hipMalloc((void**)&cam, sizeof(Camera*)));
    initCamera<<<1, 1 >>>(cam, imgTmp);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    glm::u32vec2 imgSize = glm::u32vec2(imgTmp.x, imgTmp.x / (16.0f / 9.0f)); //imGuiCam.getImageSize();

    GLFWwindow* window = glfwCreateWindow(imgSize.x, imgSize.y, "Raytracing", NULL, NULL);
    if (window == NULL)
    {
        std::cout << "Failed to create GLFW window" << std::endl;

        glfwTerminate();
        return -1;
    }

    glfwMakeContextCurrent(window);

    if (glewInit() != GLEW_OK)
        std::cout << "Error initializing GLEW" << std::endl;

    glViewport(0, 0, imgSize.x, imgSize.y);
    glfwSetFramebufferSizeCallback(window, framebuffer_size_callback);
    glEnable(GL_DEPTH_TEST);

    //imgui
    ImGui::CreateContext();
    ImGuiIO& io = ImGui::GetIO();

    ImGui_ImplGlfw_InitForOpenGL(window, true);
    ImGui_ImplOpenGL3_Init("#version 330");
    ImGui::StyleColorsDark();
    ImGui::PushStyleVar(ImGuiStyleVar_SeparatorTextAlign, ImVec2(0.5f, 0.5f));


    //Creating raytracing image plane
    Shader sh("src/Rendering/Shaders/shader.shader");

    //RAYTRACING CODE
    //To tutaj bo nie chce mi sie liczyc ile to bd :)
    int hittableCount = 1;
    for (int a = -11; a < 11; a++)
        for (int b = -11; b < 11; b++)
            hittableCount++;

    Hittable** hittableList;
    checkCudaErrors(hipMalloc((void**)&hittableList, hittableCount * sizeof(Hittable*)));
    Hittable** world;
    checkCudaErrors(hipMalloc((void**)&world, sizeof(Hittable*)));

    //Allocate randState
    uint32_t numOfChannels = 4; //RGBA
    uint32_t pixelsSize = imgSize.x * imgSize.y * numOfChannels;

    hiprandState* curRandState; //For pixels
    checkCudaErrors(hipMalloc((void**)&curRandState, pixelsSize * sizeof(hiprandState)));
    initWorld<<<1, 1>>>(world, hittableList, curRandState1);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    dataPixels* pixels;
    checkCudaErrors(hipMallocManaged((void**)&pixels, pixelsSize * sizeof(dataPixels)));

    clock_t start, stop;
    start = clock();

    // Render our buffer
    int threadsX = 8, threadsY = 8;
    dim3 blocks(imgSize.x / threadsX + 1, imgSize.y / threadsY + 1);
    dim3 threads(threadsX, threadsY);
    render_init<<<blocks, threads>>>(imgSize, curRandState);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    render<<<blocks, threads>>>(pixels, imgSize, cam, world, curRandState);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    //Texture generation
    Texture tx((unsigned char*)pixels, imgSize.x, imgSize.y);

    //DRAWING utils and preparations
    glm::vec2 size(1.0f, 1.0f);
    std::vector<uint32_t> indices = { 0,1,2, 0,2,3 };
    std::vector<float> planeVert = {
        //Position                 //Texture coords
        +size.x, +size.y, 0.0f,    1.0f, 0.0f,
        +size.x, -size.y, 0.0f,    1.0f, 1.0f,
        -size.x, -size.y, 0.0f,    0.0f, 1.0f,
        -size.x, +size.y, 0.0f,    0.0f, 0.0f,
    };

    VertexBuffer vb(planeVert.data(), planeVert.size() * sizeof(float));
    VertexArray va;
    IndexBuffer ib(indices.data(), indices.size());
    VertexBufferLayout layout;
    layout.Push<float>(3);
    layout.Push<float>(2);

    va.addBuffer(vb, layout);

    ib.unbind();
    va.unbind();
    vb.unbind();

    Renderer renderer;
    sh.unbind();

    while (!glfwWindowShouldClose(window))
    {
        float currFrame = static_cast<float>(glfwGetTime());
        deltaTime = currFrame - lastFrame;
        lastFrame = currFrame;

        renderer.clear();

        //ImGui
        ImGui_ImplOpenGL3_NewFrame();
        ImGui_ImplGlfw_NewFrame();
        ImGui::NewFrame();

        //Update
        //ImGui::ShowDemoWindow();

        processInput(window);
        sh.bind();
        sh.setUniform1i("texture1", 0);
        tx.bind(0);

        renderer.draw(va, ib, sh);

        sh.unbind();
        tx.unbind();

        //End Draw
        ImGui::Render();
        ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());

        glfwSwapBuffers(window);
        glfwPollEvents();
    }

    ImGui_ImplOpenGL3_Shutdown();
    ImGui_ImplGlfw_Shutdown();
    ImGui::DestroyContext();
    glfwTerminate();
    return 0;
}