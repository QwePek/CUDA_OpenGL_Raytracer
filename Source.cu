#include "hip/hip_runtime.h"
#include "pch.h"
#include "GL/glew.h"
#include <GLFW/glfw3.h>

#include <imgui/imgui_impl_glfw.h>
#include <imgui/imgui_impl_opengl3.h>

#include "src/Rendering/Renderer.h"
#include "src/Rendering/VertexBuffer.h"
#include "src/Rendering/VertexBufferLayout.h"
#include "src/Rendering/IndexBuffer.h"
#include "src/Rendering/Texture.h"
#include "src/Utils/Utils.h"
#include "src/Raytracing/HittableList.h"
#include "src/Raytracing/Objects/Sphere.h"
#include "src/Camera.h"
#include "src/Raytracing/Materials/Lambertian.h"
#include "src/Raytracing/Materials/Metal.h"
#include "src/Raytracing/Materials/Dielectric.h"

//CUDA
#include <hiprand/hiprand_kernel.h>
#include <>

float deltaTime = 0.0f;
float lastFrame = 0.0f;

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__global__ void render(dataPixels** data, glm::u32vec2 imgSize, Camera** cam, HittableList** world, hiprandState* rand_state)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if ((i >= imgSize.x) || (j >= imgSize.y))
        return;
    int pixelIndex = i + j * imgSize.x;
    hiprandState localRandState = rand_state[pixelIndex];
    glm::dvec3 pixelColor(0.0f, 0.0f, 0.0f);
    for (int sampleIdx = 0; sampleIdx < (*cam)->getPerPixelSamples(); sampleIdx++) {
        Ray r = (*cam)->getRay(i, j, rand_state);
        pixelColor += (*cam)->rayColor(r, (*cam)->getMaxRecursionDepth(), **world, rand_state);
    }
    rand_state[pixelIndex] = localRandState;
    *data[pixelIndex] = (*cam)->convertColor((*cam)->getPixelSampleScale() * pixelColor);
}

__global__ void rand_init(hiprandState* rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(1984, 0, 0, rand_state);
    }
}

#define RND (hiprand_uniform(&local_rand_state))

__global__ void initCamera(Camera** camera)
{
    *camera = new Camera(glm::dvec3(13, 2, 3), glm::dvec3(0, 0, 0), glm::dvec3(0, 1, 0), 20, 10.0, 0.6, 16.0f / 9.0f, 400, 20, 10);
}

__global__ void initWorld(HittableList** worldObjects, hiprandState* rand_state)
{
    if (threadIdx.x != 0 || blockIdx.x != 0)
        return;

    int hittableCount = 1;
    for (int a = -11; a < 11; a++) {
        for (int b = -11; b < 11; b++) {
            hittableCount++;
        }
    }
    int id = 0;
    (*worldObjects)->objectsSize = hittableCount;
    (*worldObjects)->objects = new Hittable*[hittableCount];
    (*worldObjects)->objects[id++] = new Sphere(glm::dvec3(0, -1000, 0), 1000, new Materials::Lambertian(glm::dvec3(0.5, 0.5, 0.5)));

    for (int a = -11; a < 11; a++) {
        for (int b = -11; b < 11; b++) {
            double choose_mat = Utils::generateRandomNumber(rand_state);
            glm::dvec3 center(a + 0.9 * Utils::generateRandomNumber(rand_state), 0.2, b + 0.9 * Utils::generateRandomNumber(rand_state));

            if ((center - glm::dvec3(4, 0.2, 0)).length() > 0.9) {
                Material* sphere_material;

                if (choose_mat < 0.8) {
                    glm::dvec3 albedo = Utils::Vector::randomVector(0.0, 1.0, rand_state) * Utils::Vector::randomVector(0.0, 1.0, rand_state);
                    sphere_material = new Materials::Lambertian(albedo);
                }
                else if (choose_mat < 0.95) {
                    glm::dvec3 albedo = Utils::Vector::randomVector(0.5, 1.0, rand_state);
                    double fuzz = Utils::generateRandomNumber(0, 0.5, rand_state);
                    sphere_material = new Materials::Metal(albedo, fuzz);
                }
                else
                    sphere_material = new Materials::Dielectric(1.5);

                (*worldObjects)->objects[id++] = new Sphere(center, 0.2, sphere_material);
            }
        }
    }
    (*worldObjects)->objects[id++] = new Sphere(glm::dvec3(0, 1, 0), 1.0, new Materials::Dielectric(1.5));
    (*worldObjects)->objects[id++] = new Sphere(glm::dvec3(-4, 1, 0), 1.0, new Materials::Lambertian(glm::dvec3(0.4, 0.2, 0.1)));
    (*worldObjects)->objects[id++] = new Sphere(glm::dvec3(4, 1, 0), 1.0, new Materials::Metal(glm::dvec3(0.7, 0.6, 0.5), 0.0));
}

__global__ void freeWorld(HittableList** worldObjects, Camera** camera) {
    delete *worldObjects;
    delete *camera;
}

void processInput(GLFWwindow* window)
{
    if (glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS)
        glfwSetWindowShouldClose(window, true);

    if (glfwGetKey(window, GLFW_KEY_X) == GLFW_PRESS)
        glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);
    else
        glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
}

void framebuffer_size_callback(GLFWwindow* window, int width, int height)
{
    glViewport(0, 0, width, height);
}

int main()
{
    if (!glfwInit())
    {
        std::cout << "Failed to initialize glfwInit()" << std::endl;
        return -1;
    }

    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

    //Essential window calculations
    hiprandState* curRandState1; //For world creation
    checkCudaErrors(hipMalloc((void**)&curRandState1, 1 * sizeof(hiprandState)));
    rand_init<<<1, 1>>>(curRandState1);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    Camera** cam;
    checkCudaErrors(hipMalloc((void**)&cam, sizeof(Camera*)));
    initCamera<<<1, 1 >>>(cam);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    //Data variables
    glm::u32vec2 imgSize = glm::dvec2(700, 800);//(*cam)->getImageSize();

    GLFWwindow* window = glfwCreateWindow(imgSize.x, imgSize.y, "Raytracing", NULL, NULL);
    if (window == NULL)
    {
        std::cout << "Failed to create GLFW window" << std::endl;

        glfwTerminate();
        return -1;
    }

    glfwMakeContextCurrent(window);

    if (glewInit() != GLEW_OK)
        std::cout << "Error initializing GLEW" << std::endl;

    glViewport(0, 0, imgSize.x, imgSize.y);
    glfwSetFramebufferSizeCallback(window, framebuffer_size_callback);
    glEnable(GL_DEPTH_TEST);

    //imgui
    ImGui::CreateContext();
    ImGuiIO& io = ImGui::GetIO();

    ImGui_ImplGlfw_InitForOpenGL(window, true);
    ImGui_ImplOpenGL3_Init("#version 330");
    ImGui::StyleColorsDark();
    ImGui::PushStyleVar(ImGuiStyleVar_SeparatorTextAlign, ImVec2(0.5f, 0.5f));


    //Creating raytracing image plane
    Shader sh("src/Rendering/Shaders/shader.shader");

    //RAYTRACING CODE
    HittableList** world;
    //To tutaj bo nie chce mi sie liczyc ile to bd :)
    int hittableCount = 1;
    for (int a = -11; a < 11; a++) {
        for (int b = -11; b < 11; b++) {
            hittableCount++;
        }
    }
    checkCudaErrors(hipMalloc((void**)&world, sizeof(HittableList*)));

    //Allocate randState
    uint32_t numOfChannels = 4; //RGBA
    uint32_t pixelsSize = imgSize.x * imgSize.y * numOfChannels;

    hiprandState* curRandState; //For pixels
    checkCudaErrors(hipMalloc((void**)&curRandState, pixelsSize * sizeof(hiprandState)));
    initWorld<<<1, 1>>>(world, curRandState1);
    checkCudaErrors(hipGetLastError());
    //checkCudaErrors(hipDeviceSynchronize());

    dataPixels** pixels;
    checkCudaErrors(hipMallocManaged((void**)&pixels, pixelsSize * sizeof(dataPixels*)));
    render<<<1, 1>>>(pixels, imgSize, cam, world, curRandState);

    //std::vector<dataPixels> pixels = cam.getPixelData();


    //Texture generation
    Texture tx((unsigned char*)pixels, imgSize.x, imgSize.y);

    //DRAWING utils and preparations
    glm::vec2 size(1.0f, 1.0f);
    std::vector<uint32_t> indices = { 0,1,2, 0,2,3 };
    std::vector<float> planeVert = {
        //Position                 //Texture coords
        +size.x, +size.y, 0.0f,    1.0f, 0.0f,
        +size.x, -size.y, 0.0f,    1.0f, 1.0f,
        -size.x, -size.y, 0.0f,    0.0f, 1.0f,
        -size.x, +size.y, 0.0f,    0.0f, 0.0f,
    };

    VertexBuffer vb(planeVert.data(), planeVert.size() * sizeof(float));
    VertexArray va;
    IndexBuffer ib(indices.data(), indices.size());
    VertexBufferLayout layout;
    layout.Push<float>(3);
    layout.Push<float>(2);

    va.addBuffer(vb, layout);

    ib.unbind();
    va.unbind();
    vb.unbind();

    Renderer renderer;
    sh.unbind();

    while (!glfwWindowShouldClose(window))
    {
        float currFrame = static_cast<float>(glfwGetTime());
        deltaTime = currFrame - lastFrame;
        lastFrame = currFrame;

        renderer.clear();

        //ImGui
        ImGui_ImplOpenGL3_NewFrame();
        ImGui_ImplGlfw_NewFrame();
        ImGui::NewFrame();

        //Update
        //ImGui::ShowDemoWindow();

        processInput(window);
        sh.bind();
        sh.setUniform1i("texture1", 0);
        tx.bind(0);

        renderer.draw(va, ib, sh);

        sh.unbind();
        tx.unbind();

        //End Draw
        ImGui::Render();
        ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());

        glfwSwapBuffers(window);
        glfwPollEvents();
    }

    ImGui_ImplOpenGL3_Shutdown();
    ImGui_ImplGlfw_Shutdown();
    ImGui::DestroyContext();
    glfwTerminate();
    return 0;
}