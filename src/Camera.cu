#include "hip/hip_runtime.h"
#include "pch.h"
#include "Camera.h"
//#include "Raytracing/Materials/Material.h"

//__device__ Camera::Camera(glm::dvec3 _lookFrom, glm::dvec3 _lookAt, glm::dvec3 _vUp, double vFOV, double _focusDist, double _defocusAngle
//    , double aspectRat, int imgWidth, int samplesPerPx, int maxDepth)
//    : focusDistance(_focusDist), defocusAngle(_defocusAngle), verticalFov(vFOV), perPixelSamples(samplesPerPx),
//    aspectRatio(aspectRat), lookFrom(_lookFrom), lookAt(_lookAt), vUp(_vUp), center(lookFrom), maxRecursionDepth(maxDepth)
//{
//    pixelSampleScale = 1.0f / perPixelSamples;
//
//    imageSize.x = imgWidth;
//    imageSize.y = imageSize.x / aspectRatio;
//    imageSize.y = (imageSize.y < 1) ? 1 : imageSize.y;
//
//    double alpha = Utils::degToRad(verticalFov);
//    double h = tan(alpha / 2);
//    glm::dvec2 viewport(1.0f, 2 * h * focusDistance);
//    viewport.x = viewport.y * (double(imageSize.x) / imageSize.y);
//
//    //Basis vectors calculate
//    w = glm::normalize(lookFrom - lookAt);
//    u = glm::normalize(glm::cross(vUp, w));
//    v = glm::cross(w, u);
//
//    glm::dvec3 viewport_u = viewport.x * u;
//    glm::dvec3 viewport_v = viewport.y * -v;
//
//    pixelDelta_u = viewport_u / (double)imageSize.x;
//    pixelDelta_v = viewport_v / (double)imageSize.y;
//
//    glm::dvec3 viewportUpperLeft = center - (focusDistance * w) - viewport_u / 2.0 - viewport_v / 2.0;
//    pixel00_loc = viewportUpperLeft + 0.5 * (pixelDelta_u + pixelDelta_v);
//
//    //Calculate defocus disk vectors
//    double defocusRadius = focusDistance * tan(Utils::degToRad(defocusAngle / 2.0));
//    defocusDisk_u = u * defocusRadius;
//    defocusDisk_v = v * defocusRadius;
//}

//__device__ glm::dvec3 Camera::rayColor(const Ray& ray, int depth, const Hittable& world, hiprandState* localRandState) const
//{
//    if (depth <= 0)
//        return glm::dvec3(0, 0, 0);
//
//	hitData data;
//	if (world.hit(ray, Interval(0.001, Utils::infinity), data)) {
//        Ray scateredRay(glm::dvec3(0.0f), glm::dvec3(0.0f));
//        glm::dvec3 attenuation;
//        if (data.mat->scatter(ray, data, attenuation, scateredRay, localRandState))
//            return attenuation * rayColor(scateredRay, depth - 1, world, localRandState);
//        
//        return glm::vec3(0.0f, 0.0f, 0.0f);;
//	}
//
//	glm::dvec3 unitDir = glm::normalize(ray.direction());
//    double a = 0.5 * (unitDir.y + 1.0);
//	//Gradient betweend white and (0.5f, 0.7f, 1.0f) color
//    return (1.0 - a) * glm::dvec3(1.0f, 1.0f, 1.0f) + a * glm::dvec3(0.5, 0.7, 1.0);
//}

//__device__ Ray Camera::getRay(int i, int j, hiprandState* localRandState) const
//{
//    glm::dvec3 offset = sampleSquare(localRandState);
//    glm::dvec3 pixelCenter = pixel00_loc + (((double)i + offset.x) * pixelDelta_u)
//        + (((double)j + offset.y) * pixelDelta_v);
//
//    glm::dvec3 rayOrigin = (defocusAngle <= 0) ? center : sampleDefocusDisk(localRandState);
//    glm::dvec3 rayDir = pixelCenter - rayOrigin;
//    
//    return Ray(rayOrigin, rayDir);
//}
//
////Camera helper functions
//__device__ glm::dvec3 Camera::sampleSquare(hiprandState* localRandState) const {
//    return glm::dvec3(Utils::generateRandomNumber(-0.5, 0.5, localRandState), 
//        Utils::generateRandomNumber(-0.5, 0.5, localRandState), 0);
//}
//
//__device__ glm::dvec3 Camera::sampleDefocusDisk(hiprandState* localRandState) const {
//    glm::dvec2 p = Utils::Vector::randomInUnitDisk(localRandState);
//    return center + (p.x * defocusDisk_u) + (p.y * defocusDisk_v);
//}
//
//__device__ inline double linearToGamma(double linearComponent) {
//    if (linearComponent > 0)
//        return sqrt(linearComponent);
//
//    return 0;
//}
//
//__device__ dataPixels Camera::convertColor(const glm::dvec3& color) {
//    static const Interval intensity(0.000, 0.999);
//
//    glm::dvec3 newColor = glm::dvec3(linearToGamma(color.r), linearToGamma(color.g), linearToGamma(color.b));
//
//    int r = int(256.0f * intensity.clamp(newColor.r));
//    int g = int(256.0f * intensity.clamp(newColor.g));
//    int b = int(256.0f * intensity.clamp(newColor.b));
//
//    dataPixels ret = { r, g, b, 255 };
//    return ret;
//}